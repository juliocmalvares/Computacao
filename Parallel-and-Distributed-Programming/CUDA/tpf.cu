#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <string>
#include <sstream>
#include <cstdio>
using namespace std;

string NumberToString (int Number){
    stringstream ss;
    ss << Number;
    return ss.str();
}

struct no{
    short x,y,z;
    bool condicao;
    //string caminho;
};

typedef no TipoChave;

struct TipoItem{
    TipoChave chave;
};

struct TipoCelula{
    TipoItem item;
    TipoCelula* prox;
};

struct TipoLista{
    TipoCelula *primeiro, *ultimo;
};

__device__ void FLVazia(TipoLista *lista){
    lista->primeiro = new TipoCelula;
    lista->ultimo = lista->primeiro;
    lista->ultimo->prox = NULL;
}

__device__ int Vazia(TipoLista lista){return (lista.primeiro == lista.ultimo);}

__device__ void Insere(TipoLista *lista, int x, int y, int z, bool condicao){
    lista->ultimo->prox = new TipoCelula;
    lista->ultimo = lista->ultimo->prox;
    lista->ultimo->item.chave.x = x;
    lista->ultimo->item.chave.y = y;
    lista->ultimo->item.chave.z = z;
    lista->ultimo->item.chave.condicao = condicao;
    //lista->ultimo->item.chave.caminho = texto;
    lista->ultimo->prox = NULL;
    //printf("Insere: %d %d %d\n", x, y, z);
}

__device__ void Retira(TipoCelula *p, TipoLista *lista, TipoItem *item){
    if(Vazia (*lista) || (p == NULL) || (p->prox == NULL)) return;

    TipoCelula *q = p->prox;
    *item = q->item;
    //printf("Retira: %d %d %d\n", item->chave.x, item->chave.y, item->chave.z);
    p->prox = q->prox;
    if(p->prox == NULL)lista->ultimo = p;
    delete q;
}

__global__ void logicaPrograma(TipoLista *lista, int xDest, int yDest, int zDest, int pi, int *contador_caminhos, int x, int y, int z){
    FLVazia(lista);
    Insere(lista, 0,0,0,false);
    //TipoCelula *aux = lista->primeiro->prox;
    int i = 0;
   // int x = 0;
    //int y = 0;
    //int z = 0;
    int soma = 0;
    bool c = false;
    //string s = "(0,0,0)";
    int contador = 0;
   // printf("%s", "Entrei");
    while(i<pi)
    {
        x = lista->primeiro->prox->item.chave.x;
        y = lista->primeiro->prox->item.chave.y;
        z = lista->primeiro->prox->item.chave.z;
        c = lista->primeiro->prox->item.chave.condicao;
        //printf("Lista 0: %d %d %d\n", lista->primeiro->item.chave.x,lista->primeiro->item.chave.y,lista->primeiro->item.chave.z);
        printf("%d %d %d %d\n", x, y, z, soma);
        if(pi == (i+1) && i!=0){break;}

        if((x != xDest) && (y != yDest) && (z != zDest))
        {
            /*Insere(lista,x+1,y,z,c,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z) + ")")); pi++;
            Insere(lista,x,y+1,z,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
            Insere(lista,x,y,z+1,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
            Insere(lista,x+1,y+1,z,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
            Insere(lista,x+1,y,z+1,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
            Insere(lista,x,y+1,z+1,true,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z+1) + ")")); pi++;
            Insere(lista,x+1,y+1,z+1,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y+1) + "," + NumberToString(z+1) + ")")); pi++;*/
            Insere(lista,x+1,y,z,c); pi++;
            Insere(lista,x,y+1,z,c); pi++;
            Insere(lista,x,y,z+1,c); pi++;
            Insere(lista,x+1,y+1,z,true); pi++;
            Insere(lista,x+1,y,z+1,true); pi++;
            Insere(lista,x,y+1,z+1,true); pi++;
            Insere(lista,x+1,y+1,z+1,true); pi++;
        }

        if((x == xDest) && (y != yDest) && (z != zDest))
        {
            /*Insere(lista,x,y+1,z,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
            Insere(lista,x,y,z+1,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
            Insere(lista,x,y+1,z+1,true,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z+1) + ")")); pi++;*/
           // printf("Condicao 1: %d %d %d %d\n", x, y, z, soma);
            Insere(lista,x,y+1,z,c); pi++;
            Insere(lista,x,y,z+1,c); pi++;
            Insere(lista,x,y+1,z+1,true); pi++;
        }

        if((x != xDest) && (y == yDest) && (z != zDest))
        {
            /*Insere(lista,x+1,y,z,c,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z) + ")")); pi++;
            Insere(lista,x,y,z+1,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
            Insere(lista,x+1,y,z+1,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;*/
            //printf("Condicao 2: %d %d %d %d\n", x, y, z, soma);
            Insere(lista,x+1,y,z,c); pi++;
            Insere(lista,x,y,z+1,c); pi++;
            Insere(lista,x+1,y,z+1,true); pi++;
        }

        if((x != xDest) && (y != yDest) && (z == zDest))
        {
           /* Insere(lista,x+1,y,z,c,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z) + ")")); pi++;
            Insere(lista,x,y+1,z,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
            Insere(lista,x+1,y+1,z,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;*/
            //printf("Condicao 3: %d %d %d %d\n", x, y, z, soma);
            Insere(lista,x+1,y,z,c); pi++;
            Insere(lista,x,y+1,z,c); pi++;
            Insere(lista,x+1,y+1,z,true); pi++;
        }

        if((x == xDest) && (y == yDest) && (z != zDest))
        {
            //Insere(lista,x,y,z+1,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
            //printf("Condicao 4: %d %d %d %d\n", x, y, z, soma);
            Insere(lista,x,y,z+1,c); pi++;
        }

        if((x == xDest) && (y != yDest) && (z == zDest))
        {
            //Insere(lista,x,y+1,z,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
            //printf("Condicao 5: %d %d %d %d\n", x, y, z, soma);
            Insere(lista,x,y+1,z,c); pi++;
        }

        if((x != xDest) && (y == yDest) && (z == zDest)){
            //Insere(lista,x+1,y,z,c,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z) + ")")); pi++;
            //printf("Condicao 6: %d %d %d %d\n", x, y, z, soma);
            Insere(lista,x+1,y,z,c); pi++;
        }
       // printf("%d %d %d %d\n", x, y, z, soma);
        if(c && (x == xDest) && (y == yDest) && (z == zDest)){
           // printf("%d %d %d %d\n", x, y, z, soma);
            contador++;
            //cout<<s<<endl<<endl;
        }

        Retira(lista->primeiro, lista, &lista->primeiro->item);
        //delete aux;
        //TipoCelula *aux = lista->primeiro->prox;
        //aux = aux->prox;

       // s = aux->item.chave.caminho;
        i++;
        soma += contador;
    }
    __syncthreads();
    *contador_caminhos = contador;

}

//Host é a CPU
//Device é a placa.

int main(){
    TipoLista *lista_device;

    int xDest_device = 1;
    int yDest_device = 2;
    int zDest_device = 3;
    int *contador_device = 0;
    int *contador_host = 0;
    int contador_final;

    //Alocação
    hipMalloc(&lista_device, sizeof(TipoLista));
    hipMalloc(&contador_device, sizeof(int));

    logicaPrograma<<<2,32>>>(lista_device, xDest_device, yDest_device, zDest_device, 1, contador_device, 0, 0, 0);
    hipMemcpy(&contador_host, &contador_device, sizeof(int*), hipMemcpyDeviceToHost);
    hipFree(lista_device);
    hipFree(contador_device);

    stringstream stream;
    stream << contador_host;
    stream >> std::hex >> contador_final;

    cout << "Caminhos:" << contador_final << endl;

    return 0;
}


