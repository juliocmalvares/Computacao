#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <string>
#include <ctime>
#include <omp.h>
#include <cstdio>
#include <unistd.h>
#include <iomanip>
#include <hip/hip_runtime.h>

using namespace std;

__device__ string NumberToString (int Number){
    stringstream ss;
    ss << Number;
    return ss.str();
}

struct no{
    short x,y,z;
    bool condicao;
    string caminho;
};

typedef no TipoChave;

struct TipoItem{
    TipoChave chave;
};

struct TipoCelula{
    TipoItem item;
    TipoCelula* prox;
};

struct TipoLista{
    TipoCelula *primeiro, *ultimo;
};

__device__ void FLVazia(TipoLista *lista){
    lista->primeiro = new TipoCelula;
    lista->ultimo = lista->primeiro;
    lista->ultimo->prox = NULL;
}

__device__ int Vazia(TipoLista lista){return (lista.primeiro == lista.ultimo);}

__device__ void Insere(TipoLista *lista, int x,int y,int z,bool condicao, string s){
    lista->ultimo->prox = new TipoCelula;
    lista->ultimo = lista->ultimo->prox;
    lista->ultimo->item.chave.x = x;
    lista->ultimo->item.chave.y = y;
    lista->ultimo->item.chave.z = z;
    lista->ultimo->item.chave.condicao = condicao;
    //cout<<s<<endl;
    lista->ultimo->item.chave.caminho = s;
    //cout<<lista->ultimo->item.chave.caminho<<endl;
    lista->ultimo->prox = NULL;
}

__device__ void Retira(TipoCelula *p, TipoLista *lista, TipoItem *item){
    if(Vazia (*lista) || (p == NULL) || (p->prox == NULL)) return;

    TipoCelula *q = p->prox;
    *item = q->item;

    p->prox = q->prox;

    if(p->prox == NULL)lista->ultimo = p;
    delete q;
}

__global__ void logicaPrograma(TipoLista *lista,int xDest, int yDest, int zDest, int pi){
       TipoCelula *aux = lista->primeiro->prox;
       int i = 0;
       int x = 0;
       int y = 0;
       int z = 0;
       string s = "(0,0,0)";
       bool c = false;
       int contador = 0;
       int soma = 0;

       //TipoCelula *aux = lista->primeiro->prox;
   	    while(i<pi){
   	    
   	        if(((pi == (i+1) && i!=0 && (lista->primeiro->prox != NULL))))break;
   		else{
   			if(lista->primeiro->prox == NULL || x>xDest || y>yDest || z>zDest){break;}
   		}
   				x = lista->primeiro->prox->item.chave.x;		        
   				y = lista->primeiro->prox->item.chave.y;
   		        z = lista->primeiro->prox->item.chave.z;
   		        c = lista->primeiro->prox->item.chave.condicao;
   				s = lista->primeiro->prox->item.chave.caminho;
   				//
   			

   			//cout<<endl;
   			//cout<<"("<<x<<","<<y<<","<<z<<")"<<omp_get_thread_num()<<endl;
   			if((x != xDest) && (y != yDest) && (z != zDest)){
   			    Insere(lista,x+1,y,z,c,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z) + ")")); pi++;			    //cout<<(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z) + ")");
   		            Insere(lista,x,y+1,z,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
   		            Insere(lista,x,y,z+1,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
   		            Insere(lista,x+1,y+1,z,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
   		            Insere(lista,x+1,y,z+1,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
   		            Insere(lista,x,y+1,z+1,true,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z+1) + ")")); pi++;
   		            Insere(lista,x+1,y+1,z+1,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y+1) + "," + NumberToString(z+1) + ")")); pi++;
   		        }

   		        if((x == xDest) && (y != yDest) && (z != zDest)){
   			    Insere(lista,x,y+1,z,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
   		            Insere(lista,x,y,z+1,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
   		            Insere(lista,x,y+1,z+1,true,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z+1) + ")")); pi++;
   		        }

   		        if((x != xDest) && (y == yDest) && (z != zDest)){
   		            Insere(lista,x+1,y,z,c,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z) + ")")); pi++;
   		            Insere(lista,x,y,z+1,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
   		            Insere(lista,x+1,y,z+1,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
   		        }

   		        if((x != xDest) && (y != yDest) && (z == zDest)){
   		            Insere(lista,x+1,y,z,c,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z) + ")")); pi++;
   		            Insere(lista,x,y+1,z,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
   		            Insere(lista,x+1,y+1,z,true,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
   		        }

   		        if((x == xDest) && (y == yDest) && (z != zDest)){
   		            Insere(lista,x,y,z+1,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y) + "," + NumberToString(z+1) + ")")); pi++;
   		        }

   		        if((x == xDest) && (y != yDest) && (z == zDest)){
   		            Insere(lista,x,y+1,z,c,(s + "->("+ NumberToString(x) + "," + NumberToString(y+1) + "," + NumberToString(z) + ")")); pi++;
   		        }

   		        if((x != xDest) && (y == yDest) && (z == zDest)){
   		            Insere(lista,x+1,y,z,c,(s + "->("+ NumberToString(x+1) + "," + NumberToString(y) + "," + NumberToString(z) + ")")); pi++;
   		        }

   		        if(c && (x == xDest) && (y == yDest) && (z == zDest)){
   		            contador++;
   			    //cout<<s<<NumberToString(x+1);
   		            cout<<s<<endl<<endl;
   		        }

   			Retira(lista->primeiro, lista, &lista->primeiro->item);        	
   		        //delete aux;
   		
   	        i++;
   	    }
       

    //return contador;
}

double calcula_resultado(clock_t fim, clock_t inicio){
	return (fim - inicio)/(double)CLOCKS_PER_SEC;
}

int main(){
    
    TipoLista lista;
    FLVazia(&lista);
    unsigned long long int pi = 1;
    TipoItem item;
    int xDest = 5;
    int yDest = 5;
    int zDest = 5;
    int contador;
    double time;
    
    clock_t fim, inicio;
    double tempo;
    inicio = clock();
    Insere(&lista,0,0,0,false,"0,0,0");
    contador = logicaPrograma(&lista, xDest, yDest, zDest, pi);
    fim = clock();

    cout<<endl<<"Numero de caminhos: "<< contador<<endl;
    time = calcula_resultado(fim, inicio);
    cout << "Tempo de execução: " << calcula_resultado(fim, inicio) << "ms" <<endl;


    FILE *f;
    f = fopen("exec-parallel-555.txt", "w");
    fprintf(f, "%.6f segundos\n", time);
    fprintf(f, "%d caminhos \n", contador);

    return 0;
}

